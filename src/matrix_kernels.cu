#include "hip/hip_runtime.h"
#include "matrix.hpp"

__global__ void matmul_k(float* lhs, float* rhs, float* res, size_t rows,
                         size_t mid, size_t cols) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < rows && col < cols) {
    float sum = 0;
    for (int i = 0; i < mid; i++) {
      sum += lhs[row * mid + i] * rhs[i * cols + col];
    }
    res[row * cols + col] = sum;
  }
}

void matmul_wrapper(float* lhs, float* rhs, float* res, size_t rows, size_t mid,
                    size_t cols) {
  float *d_l, *d_r, *d_res;
  hipMalloc(&d_l, sizeof(float) * rows * mid);
  hipMalloc(&d_r, sizeof(float) * mid * cols);
  hipMalloc(&d_res, sizeof(float) * rows * cols);

  hipMemcpy(d_l, lhs, sizeof(float) * rows * mid, hipMemcpyHostToDevice);
  hipMemcpy(d_r, rhs, sizeof(float) * mid * cols, hipMemcpyHostToDevice);

  uint grid_x = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
  uint grid_y = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 gridsize(grid_x, grid_y);
  dim3 blocksize(BLOCK_SIZE, BLOCK_SIZE);

  matmul_k<<<gridsize, blocksize>>>(d_l, d_r, d_res, rows, mid, cols);
  hipDeviceSynchronize();

  hipMemcpy(res, d_res, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);
  hipFree(d_l);
  hipFree(d_r);
  hipFree(d_res);
}

__global__ void matadd_k(float* lhs, float* rhs, size_t rows, size_t cols,
                         bool sub, bool broadcast) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < rows && col < cols) {
    lhs[row * cols + col] +=
        (sub ? -1 : 1) * (broadcast ? rhs[row] : rhs[row * cols + col]);
  }
}

void matadd_wrapper(float* lhs, float* rhs, size_t rows, size_t cols, bool sub,
                    bool broadcast) {
  float *d_l, *d_r;
  hipMalloc(&d_l, sizeof(float) * rows * cols);
  hipMalloc(&d_r, sizeof(float) * rows * (broadcast ? 1 : cols));

  hipMemcpy(d_l, lhs, sizeof(float) * rows * cols, hipMemcpyHostToDevice);
  hipMemcpy(d_r, rhs, sizeof(float) * rows * (broadcast ? 1 : cols),
             hipMemcpyHostToDevice);

  uint grid_x = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
  uint grid_y = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 gridsize(grid_x, grid_y);
  dim3 blocksize(BLOCK_SIZE, BLOCK_SIZE);

  matadd_k<<<gridsize, blocksize>>>(d_l, d_r, rows, cols, sub, broadcast);
  hipDeviceSynchronize();

  hipMemcpy(lhs, d_l, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);
  hipFree(d_l);
  hipFree(d_r);
}

__global__ void mataddscal_k(float* lhs, float rhs, size_t rows, size_t cols,
                             bool sub) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < rows && col < cols) {
    lhs[row * cols + col] += (sub ? -1 : 1) * rhs;
  }
}

void mataddscal_wrapper(float* lhs, float rhs, size_t rows, size_t cols,
                        bool sub) {
  float* d_l;
  hipMalloc(&d_l, sizeof(float) * rows * cols);

  hipMemcpy(d_l, lhs, sizeof(float) * rows * cols, hipMemcpyHostToDevice);

  uint grid_x = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
  uint grid_y = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 gridsize(grid_x, grid_y);
  dim3 blocksize(BLOCK_SIZE, BLOCK_SIZE);

  mataddscal_k<<<gridsize, blocksize>>>(d_l, rhs, rows, cols, sub);
  hipDeviceSynchronize();

  hipMemcpy(lhs, d_l, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);
  hipFree(d_l);
}