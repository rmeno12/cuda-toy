#include "matrix.hpp"
#include "matrix_kernels.cuh"

void matmul_wrapper(float* lhs, float* rhs, float* res, size_t rows, size_t mid,
                    size_t cols) {
  float *d_l, *d_r, *d_res;
  hipMalloc(&d_l, sizeof(float) * rows * mid);
  hipMalloc(&d_r, sizeof(float) * mid * cols);
  hipMalloc(&d_res, sizeof(float) * rows * cols);

  hipMemcpy(d_l, lhs, sizeof(float) * rows * mid, hipMemcpyHostToDevice);
  hipMemcpy(d_r, rhs, sizeof(float) * mid * cols, hipMemcpyHostToDevice);

  uint grid_x = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
  uint grid_y = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 gridsize(grid_x, grid_y);
  dim3 blocksize(BLOCK_SIZE, BLOCK_SIZE);

  matmul_k<<<gridsize, blocksize>>>(d_l, d_r, d_res, rows, mid, cols);
  hipDeviceSynchronize();

  hipMemcpy(res, d_res, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);
  hipFree(d_l);
  hipFree(d_r);
  hipFree(d_res);
}

void matadd_wrapper(float* lhs, float* rhs, size_t rows, size_t cols, bool sub,
                    bool broadcast) {
  float *d_l, *d_r;
  hipMalloc(&d_l, sizeof(float) * rows * cols);
  hipMalloc(&d_r, sizeof(float) * rows * (broadcast ? 1 : cols));

  hipMemcpy(d_l, lhs, sizeof(float) * rows * cols, hipMemcpyHostToDevice);
  hipMemcpy(d_r, rhs, sizeof(float) * rows * (broadcast ? 1 : cols),
             hipMemcpyHostToDevice);

  uint grid_x = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
  uint grid_y = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 gridsize(grid_x, grid_y);
  dim3 blocksize(BLOCK_SIZE, BLOCK_SIZE);

  matadd_k<<<gridsize, blocksize>>>(d_l, d_r, rows, cols, sub, broadcast);
  hipDeviceSynchronize();

  hipMemcpy(lhs, d_l, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);
  hipFree(d_l);
  hipFree(d_r);
}

void matprod_wrapper(float* lhs, float* rhs, size_t rows, size_t cols) {
  float *d_l, *d_r;
  hipMalloc(&d_l, sizeof(float) * rows * cols);
  hipMalloc(&d_r, sizeof(float) * rows * cols);

  hipMemcpy(d_l, lhs, sizeof(float) * rows * cols, hipMemcpyHostToDevice);
  hipMemcpy(d_r, rhs, sizeof(float) * rows * cols, hipMemcpyHostToDevice);

  uint grid_x = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
  uint grid_y = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 gridsize(grid_x, grid_y);
  dim3 blocksize(BLOCK_SIZE, BLOCK_SIZE);

  matprod_k<<<gridsize, blocksize>>>(d_l, d_r, rows, cols);
  hipDeviceSynchronize();

  hipMemcpy(lhs, d_l, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);
  hipFree(d_l);
  hipFree(d_r);
}

void matdiv_wrapper(float* lhs, float* rhs, size_t rows, size_t cols) {
  float *d_l, *d_r;
  hipMalloc(&d_l, sizeof(float) * rows * cols);
  hipMalloc(&d_r, sizeof(float) * rows * cols);

  hipMemcpy(d_l, lhs, sizeof(float) * rows * cols, hipMemcpyHostToDevice);
  hipMemcpy(d_r, rhs, sizeof(float) * rows * cols, hipMemcpyHostToDevice);

  uint grid_x = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
  uint grid_y = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 gridsize(grid_x, grid_y);
  dim3 blocksize(BLOCK_SIZE, BLOCK_SIZE);

  matdiv_k<<<gridsize, blocksize>>>(d_l, d_r, rows, cols);
  hipDeviceSynchronize();

  hipMemcpy(lhs, d_l, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);
  hipFree(d_l);
  hipFree(d_r);
}

void mataddscal_wrapper(float* lhs, float rhs, size_t rows, size_t cols,
                        bool sub) {
  float* d_l;
  hipMalloc(&d_l, sizeof(float) * rows * cols);

  hipMemcpy(d_l, lhs, sizeof(float) * rows * cols, hipMemcpyHostToDevice);

  uint grid_x = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
  uint grid_y = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 gridsize(grid_x, grid_y);
  dim3 blocksize(BLOCK_SIZE, BLOCK_SIZE);

  mataddscal_k<<<gridsize, blocksize>>>(d_l, rhs, rows, cols, sub);
  hipDeviceSynchronize();

  hipMemcpy(lhs, d_l, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);
  hipFree(d_l);
}

void matprodscal_wrapper(float* lhs, float rhs, size_t rows, size_t cols) {
  float* d_l;
  hipMalloc(&d_l, sizeof(float) * rows * cols);

  hipMemcpy(d_l, lhs, sizeof(float) * rows * cols, hipMemcpyHostToDevice);

  uint grid_x = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
  uint grid_y = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 gridsize(grid_x, grid_y);
  dim3 blocksize(BLOCK_SIZE, BLOCK_SIZE);

  matprodscal_k<<<gridsize, blocksize>>>(d_l, rhs, rows, cols);
  hipDeviceSynchronize();

  hipMemcpy(lhs, d_l, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);
  hipFree(d_l);
}

void matdivscal_wrapper(float* lhs, float rhs, size_t rows, size_t cols) {
  float* d_l;
  hipMalloc(&d_l, sizeof(float) * rows * cols);

  hipMemcpy(d_l, lhs, sizeof(float) * rows * cols, hipMemcpyHostToDevice);

  uint grid_x = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
  uint grid_y = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 gridsize(grid_x, grid_y);
  dim3 blocksize(BLOCK_SIZE, BLOCK_SIZE);

  matdivscal_k<<<gridsize, blocksize>>>(d_l, rhs, rows, cols);
  hipDeviceSynchronize();

  hipMemcpy(lhs, d_l, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);
  hipFree(d_l);
}

void mattranspose_wrapper(float* mat, float* res, size_t rows, size_t cols) {
  float *d_mat, *d_res;
  hipMalloc(&d_mat, sizeof(float) * rows * cols);
  hipMalloc(&d_res, sizeof(float) * rows * cols);

  hipMemcpy(d_mat, mat, sizeof(float) * rows * cols, hipMemcpyHostToDevice);
  hipMemcpy(d_res, res, sizeof(float) * rows * cols, hipMemcpyHostToDevice);

  uint grid_x = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
  uint grid_y = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 gridsize(grid_x, grid_y);
  dim3 blocksize(BLOCK_SIZE, BLOCK_SIZE);

  mattranspose_k<<<gridsize, blocksize>>>(d_mat, d_res, rows, cols);
  hipDeviceSynchronize();

  hipMemcpy(res, d_res, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);
  hipFree(d_mat);
  hipFree(d_res);
}